
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

int main() {
  float *inputs, *weights1, *bias1, *weights2, *bias2, *weights3, *bias3, *output;

  hipMallocManaged(&inputs, 4*sizeof(float));
  hipMallocManaged(&weights1, 4*sizeof(float));
  hipMallocManaged(&bias1, sizeof(float));
  hipMallocManaged(&weights2, 4*sizeof(float));
  hipMallocManaged(&bias2, sizeof(float));
  hipMallocManaged(&weights3, 4*sizeof(float));
  hipMallocManaged(&bias3, sizeof(float));
  hipMallocManaged(&output, 3*sizeof(float));

  inputs[0] = 1.0f;
  inputs[1] = 2.0f;
  inputs[2] = 3.0f;
  inputs[3] = 2.5f;

  weights1[0] = 0.2f;
  weights1[1] = 0.8f;
  weights1[2] = -0.5f;
  weights1[3] = 1.0f;
  weights2[0] = 0.5f;
  weights2[1] = -0.91f;
  weights2[2] = 0.26f;
  weights2[3] = -0.5f;
  weights3[0] = -0.26f;
  weights3[1] = -0.27f;
  weights3[2] = 0.17f;
  weights3[3] = 0.87f;

  bias1[0] = 2.0f;
  bias2[0] = 3.0f;
  bias3[0] = 0.5f;

  output[0] = inputs[0]*weights1[0] + inputs[1]*weights1[1] + inputs[2]*weights1[2] + bias1[0];
  output[1] = inputs[0]*weights2[0] + inputs[1]*weights2[1] + inputs[2]*weights2[2] + bias2[0];
  output[2] = inputs[0]*weights3[0] + inputs[1]*weights3[1] + inputs[2]*weights3[2] + bias3[0];

  std::cout << "[" << output[0] << ", " << output[1] << ", " << output[2] << "]" << std::endl;

  hipFree(inputs);
  hipFree(weights1);
  hipFree(bias1);
  hipFree(weights2);
  hipFree(bias2);
  hipFree(weights3);
  hipFree(bias3);
  hipFree(output);
}
